#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 1024

__global__ void reduce_sum(float* input, float* output) {
    // Allocate shared memory for the block
    extern __shared__ float sdata[];
    
    // Each thread loads one element from global to shared memory
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    sdata[tid] = input[i];
    __syncthreads();
    
    // Perform reduction in shared memory
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }
    
    // Write result for this block to global memory
    if (tid == 0) {
        output[blockIdx.x] = sdata[0];
    }
}

int main() {
    float *h_input, *h_output; // host variables
    float *d_input, *d_output; // device variables

    size_t size = N * sizeof(float);
    h_input = (float*) malloc(size);
    h_output = (float*) malloc(sizeof(float));

    for (int i = 0; i < N; i++) {
        h_input[i] = 1.0f;
    }

    hipMalloc(&d_input, size);
    hipMalloc(&d_output, sizeof(float));
    hipMemcpy(d_input, h_input, size, hipMemcpyHostToDevice);

    reduce_sum<<<1, N, N * sizeof(float)>>>(d_input, d_output);

    hipMemcpy(h_output, d_output, sizeof(float), hipMemcpyDeviceToHost);

    printf("Sum = %f\n", h_output[0]);

    free(h_input);
    free(h_output);
    hipFree(d_input);
    hipFree(d_output);
    return 0;
}
